
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;
using namespace std::chrono;

#define MAX_THREADS 2
#define MAX_BLOCKS 8

float** h_A;
float* h_csr_vals;
int* h_csr_rows;
int* h_csr_col;
float* h_B;
float* h_C1;
float* h_C2;
int n, m;

//sequential multiplication
void sequentialMulti(){
	for(int i=0; i<n; i++)
    {
        h_C1[i] = 0;
		for(int k = h_csr_rows[i]; k < h_csr_rows[i+1]; k++)
        {
            int index = h_csr_col[k];
            h_C1[i]+= (h_csr_vals[k]*h_B[index]);
        }
    }
}

__global__ void parallelMulti(int d_n, int* d_csr_rows, float* d_C2, float* d_csr_vals, float* d_B, int* d_csr_col){

	long long int thread_id = blockIdx.x * MAX_THREADS + threadIdx.x;
	int m = d_n;
	int lx = (thread_id*(d_n+m+1))/MAX_THREADS;
	int rx = (thread_id+1)*(d_n+m+1); rx/=MAX_THREADS;
	
	int l = 0, r = lx;
    int r_el_f_A = 0;
    
    while(l<=r){
        int mid = l+(r-l)/2;
        int x = mid -1;
        int y = lx - mid - 1;
        
        if(x>d_n)
            r = mid-1;
        else if(x<0 || y<0 || d_csr_rows[x]<=y)
            r_el_f_A = mid,
            l = mid + 1;
        else
            r = mid -1;
    }
	
	int ind_el_A = lx - r_el_f_A;
	int r_el_f_B = d_n+1;
	if(thread_id!=MAX_THREADS-1)
 	{
		int l = 0, r = rx;
		r_el_f_B = 0;
		
		while(l<=r){
			int mid = l+(r-l)/2;
			int x = mid -1;
			int y = rx - mid - 1;
			
			if(x>d_n)
				r = mid-1;
			else if(x<0 || y<0 || d_csr_rows[x]<=y)
				r_el_f_B = mid,
				l = mid + 1;
			else
				r = mid -1;
		}
	 }

	int ind_el_B = rx - r_el_f_B;
	
	int i = r_el_f_A; int j = ind_el_A;
	int N = r_el_f_B; int M = ind_el_B;
	
	while(i<N && j<M){
	    if(d_csr_rows[i]<=j) //no entries in the row (moving down).
	        i++;
	    else{
	       //moving right and adding row_element*corresponding_element_of_vector to answer vector.
	       d_C2[i-1]+= d_csr_vals[j] * d_B[d_csr_col[j]];
	       j++;
	   }
	}
	while(i<N){
	    i++;
	}
	while(j<M){
	    d_C2[i-1]+= d_csr_vals[j] * d_B[d_csr_col[j]];
	    j++;
	}
}

int main(){
	auto start = high_resolution_clock::now();
	
	vector<double> vals;
	vector<int> rows;
	vector<int> col;

    
	//reading inputfile.mtx
	ifstream file("inputfile.mtx");
	
	//removing header comments
	while(file.peek() == '%')
	file.ignore(1000, '\n');

	int x, y;
	double z;
	int r, c, nz;
	
	//reading number of rows, cols, and non-zero values.
	file>>r>>c>>nz;
	n=r;
	m=c;
	//A.resize(r, vector<double>(c, 0.0));

	//storing data in matrix
	for(int i = 0; i < nz; i++){
		file>>x>>y>>z;
		rows.push_back(x);
		col.push_back(y);
		vals.push_back(z);
		//A[x-1][y-1] = z;
	}
	file.close();
	
	//converting coo format to csr with 3 arrays: values, columns and number of non zero values till a particular row

	h_csr_vals = (float*)malloc(nz*sizeof(float));
	h_csr_col = (int*)malloc(c*sizeof(int));
	h_csr_rows = (int*)malloc((r+1)*sizeof(int));
	h_B = (float*)malloc(r*sizeof(float));
	for(int i = 0; i < nz; i++){
		h_csr_vals[i] = vals[i];
		h_csr_col[i] = col[i]-1;
		h_csr_rows[rows[i]]++;
	}
	for(int i = 0; i < r; i++)
	h_csr_rows[i+1] += h_csr_rows[i];
	
	// reading vector.txt
	ifstream file1("vector.txt");
	int val_B; int p = 0;
	while(file1>>val_B){
		h_B[p] = val_B;
		file1.ignore(1,' ');
		p++;
	}
	file1.close();


	h_C1 = (float*)malloc(r*sizeof(float));
	h_C2 = (float*)malloc(r*sizeof(float));

	
	//A*B sequentially
	sequentialMulti();
	cout<<endl;


	//Multi-threading (A*B in parallel using CUDA)
	//Memory allocation in GPU
	int* d_csr_rows;
	hipMalloc(&d_csr_rows, (n+1)*sizeof(int));
	float* d_C2;
	hipMalloc(&d_C2, n*sizeof(float));
	float* d_csr_vals;
	hipMalloc(&d_csr_vals, n*sizeof(float));
	float* d_B;
	hipMalloc(&d_B, n*sizeof(float));
	int* d_csr_col;
	hipMalloc(&d_csr_col, n*sizeof(int));

	hipMemcpy(d_csr_rows, h_csr_rows, (n+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csr_col, h_csr_col, (n)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, (n)*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_csr_vals, h_csr_vals, (n)*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C2, h_C2, (n)*sizeof(float), hipMemcpyHostToDevice);
    
	//SpMV using CUDA
	parallelMulti<<<MAX_BLOCKS, MAX_THREADS>>>(n, d_csr_rows, d_C2, d_csr_vals, d_B, d_csr_col);
	
	hipMemcpy(h_C2, d_C2, n*sizeof(float), hipMemcpyDeviceToHost);
	
	//Free the memory allocated in GPU
	hipFree(d_csr_rows);
	hipFree(d_csr_vals);
	hipFree(d_csr_col);
	hipFree(d_B);
	hipFree(d_C2);

	//printing sequential answer
	cout<<"Sequential Answer: \n";
	for(int i = 0; i < r; i++)
	cout<<h_C1[i]<<" ";
	cout<<endl;
	cout<<endl;

	//printing final answer using CUDA
	cout<<"Final Answer using CUDA: \n";
	for(int i = 0; i < r; i++)
	cout<<h_C2[i]<<" ";
	cout<<endl;

	
	//for execution time calculation
	cout<<endl;
	auto stop = high_resolution_clock::now();
	float duration = duration_cast<nanoseconds>(stop - start).count();
	duration*= 1e-9;
	cout << "Time taken using "<< MAX_BLOCKS<<" blocks and "<<MAX_THREADS <<" threads is " << fixed 
         << duration << setprecision(9);
    cout << " sec" << endl;
}
